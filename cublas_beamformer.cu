#include "hip/hip_runtime.h"
////#include <stdio.h>
////#include <stdlib.h>
////#include <cstdlib>
////#include <hiprand.h>
////#include <assert.h>
////#include <unistd.h>
////#include <hipblas.h>
////#include <iostream>
////#include <complex.h>
////#include <math.h>
////#include <hip/hip_complex.h>
////#include <hip/hip_runtime.h>
////#include "cublas_beamformer.h"
////
////using namespace std;
////
////// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
////void GPU_fill(hipComplex *A, int nr_rows_A, int nr_cols_A) {
////
////	/*
////	// Create a pseudo-random number generator
////	hiprandGenerator_t prng;
////	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
////
////	// Set the seed for the random number generator using the system clock
////	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());
////
////	// Fill the array with random numbers on the device
////	hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
////	*/
//////	float *G;
////
////	hipComplex *G;
////	G = new hipComplex[nr_rows_A*nr_cols_A];
////	for(int i = 0; i < nr_rows_A*nr_cols_A; i++){
////		G[i].x = 1;
////		G[i].y = 0;
////	}
////	hipMemcpy(A,G,nr_rows_A * nr_cols_A * sizeof(hipComplex),hipMemcpyHostToDevice);
////	delete[] G;
////}
////
////
//////Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
////void print_matrix(const hipComplex *A, int nr_rows_A, int nr_cols_A) {
////	for(int i = 0; i < nr_rows_A; ++i){
////		for(int j = 0; j < nr_cols_A; ++j){
////			std::cout << A[j * nr_rows_A + i].x + A[j * nr_rows_A + i].y << " ";
////		}
////		std::cout << std::endl;
////	}
////	std::cout << std::endl;
////}
////
//////void beamform(const hipComplex * h_A,
//////	const hipComplex * h_B,
//////	hipComplex * h_C) { {
////int main(){
////	// Allocate 3 arrays on CPU
////	hipError_t cudaStat;
////
////	int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;
////
//// 	nr_rows_A = N_TIME_STI;
//// 	nr_cols_A = N_ELE;
//// 	nr_rows_B = N_ELE;
//// 	nr_cols_B = N_BEAM;
//// 	nr_rows_C = N_TIME_STI;
//// 	nr_cols_C = N_BEAM;
////
////	// for simplicity we are going to use square arrays
////	//nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = 3;
////
//// 	hipComplex *h_A = (hipComplex *)malloc(nr_rows_A * nr_cols_A * N_STI * N_BIN * sizeof(hipComplex));
//// 	hipComplex *h_B = (hipComplex *)malloc(nr_rows_B * nr_cols_B * N_STI * N_BIN * sizeof(hipComplex));
//// 	hipComplex *h_C = (hipComplex *)malloc(nr_rows_C * nr_cols_C * N_STI * N_BIN * sizeof(hipComplex));
////
////	// Allocate 3 arrays on GPU
//// 	hipComplex *d_A, *d_B, *d_C;
////	hipMalloc(&d_A,nr_rows_A * nr_cols_A * N_STI * N_BIN * sizeof(hipComplex));
////	hipMalloc(&d_B,nr_rows_B * nr_cols_B * N_STI * N_BIN * sizeof(hipComplex));
////	hipMalloc(&d_C,nr_rows_C * nr_cols_C * N_STI * N_BIN * sizeof(hipComplex));
////
////	// Fill the arrays A and B on GPU with random numbers
////	GPU_fill(d_A, nr_rows_A*N_STI, nr_cols_A*N_BIN);
////	GPU_fill(d_B, nr_rows_B*N_STI, nr_cols_B*N_BIN);
////
////	// Optionally we can copy the data back on CPU and print the arrays
////	hipMemcpy(h_A,d_A,nr_rows_A * nr_cols_A * N_STI * N_BIN * sizeof(hipComplex),hipMemcpyDeviceToHost);
////	hipMemcpy(h_B,d_B,nr_rows_B * nr_cols_B * N_STI * N_BIN * sizeof(hipComplex),hipMemcpyDeviceToHost);
//////	std::cout << "A =" << std::endl;
//////	print_matrix(h_A, nr_rows_A*N_STI, nr_cols_A*N_BIN);
//////	std::cout << "B =" << std::endl;
//////	print_matrix(h_B, nr_rows_B*N_STI, nr_cols_B*N_BIN);
////
////	hipComplex **h_arr_A = 0; hipComplex **h_arr_B = 0; hipComplex **h_arr_C = 0;
////	//New code ////////
////	h_arr_A = (hipComplex **)malloc(nr_rows_A * nr_cols_A *N_STI*N_BIN*sizeof(hipComplex*));
////	h_arr_B = (hipComplex **)malloc(nr_rows_B * nr_cols_B *N_STI*N_BIN*sizeof(hipComplex*));
////	h_arr_C = (hipComplex **)malloc(nr_rows_C * nr_cols_C *N_STI*N_BIN*sizeof(hipComplex*));
////
////	for(int i = 0; i < N_STI*N_BIN; i++){
////		h_arr_A[i] = d_A + i*N_ELE*N_TIME_STI;
////		h_arr_B[i] = d_B + i*N_ELE*N_BEAM;
////		h_arr_C[i] = d_C + i*N_TIME_STI*N_BEAM;
////	}
////
//////	float At[40*38];
//////	hipMemcpy(At,h_arr_B[1],38*7*sizeof(float),hipMemcpyDeviceToHost);
//////	std::cout << "At =" << std::endl;
//////	for(int i =0; i<38*7;i++)
//////		std::cout << i << ":\t" << At[i] << std::endl;
////
////	hipComplex **d_arr_A = 0; hipComplex **d_arr_B = 0; hipComplex **d_arr_C = 0;
////	cudaStat = hipMalloc(&d_arr_A,nr_rows_A * nr_cols_A * N_STI * N_BIN * sizeof(hipComplex*));
////	assert(!cudaStat);
////	cudaStat = hipMalloc(&d_arr_B,nr_rows_B * nr_cols_B * N_STI * N_BIN * sizeof(hipComplex*));
////	assert(!cudaStat);
////	cudaStat = hipMalloc(&d_arr_C,nr_rows_C * nr_cols_C * N_STI * N_BIN * sizeof(hipComplex*));
////	assert(!cudaStat);
////
////	//hipMemcpy(d_arr_A,h_arr_A[0],nr_rows_A * nr_cols_A * N_STI * N_BIN * sizeof(float*),hipMemcpyHostToDevice);
////
////	//printf("H_arr_A %d",&h_arr_A[0]);
////
////	cudaStat = hipMemcpy(d_arr_A,h_arr_A,nr_rows_A * nr_cols_A * N_STI * N_BIN * sizeof(hipComplex*),hipMemcpyHostToDevice);
////	assert(!cudaStat);
////	cudaStat = hipMemcpy(d_arr_B,h_arr_B,nr_rows_B * nr_cols_B * N_STI * N_BIN * sizeof(hipComplex*),hipMemcpyHostToDevice);
////	assert(!cudaStat);
////	cudaStat = hipMemcpy(d_arr_C,h_arr_C,nr_rows_C * nr_cols_C * N_STI * N_BIN * sizeof(hipComplex*),hipMemcpyHostToDevice);
////	assert(!cudaStat);
////	// Multiply A and B on GPU
////
////	//gpu_blas_mmul(d_arr_A, d_arr_B, d_arr_C, nr_rows_A, nr_rows_B, nr_cols_A);
////	int lda=nr_rows_A,ldb=nr_rows_B,ldc=nr_rows_A;
////	hipComplex alf;
////	hipComplex bet;
////
////	alf.x = 1;
////	alf.y = 0;
////	bet.x = 0;
////	bet.y = 0;
//////	const float *alpha = &alf;
//////	const float *beta = &bet;
////	//New variables
////	int batchCount = N_STI*N_BIN;
////
////	// Create a handle for CUBLAS
////	hipblasHandle_t handle;
////	hipblasCreate(&handle);
////
////	hipblasStatus_t stat;
////	// Do the actual multiplication
//////	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
////
////	stat = hipblasCgemmBatched(
////			handle,
////			HIPBLAS_OP_N,
////			HIPBLAS_OP_N,
////			nr_rows_A,
////			nr_cols_A,
////			nr_rows_B,
////			&alf,
////			(const hipComplex **)d_arr_A,
////			lda,
////			(const hipComplex **)d_arr_B,
////			ldb,
////			&bet,
////			(hipComplex **)d_arr_C,
////			ldc,
////			batchCount);
////
////
////	if(stat != HIPBLAS_STATUS_SUCCESS){
////		cerr << "hipblasSgemmBatched failed" << endl;
////		exit(1);
////	}
////	assert(!hipGetLastError());
////
////	//////////////////////
////
////
////	// Multiply A and B on GPU
////
////	//gpu_blas_mmul(d_A, d_B, d_C, nr_rows_A, nr_rows_B, nr_cols_A);
////
////	// Copy (and print) the result on host memory
////	hipMemcpy(h_C,h_arr_C[0],nr_rows_C * nr_cols_C * N_STI * N_BIN* sizeof(hipComplex),hipMemcpyDeviceToHost); //d_c => h_arr_C[0]
////
////	std::cout << "C =" << std::endl;
////	print_matrix(h_C, nr_rows_C*N_STI, nr_cols_C*N_BIN);
////
////	//Free GPU memory
////	hipFree(d_A);
////	hipFree(d_B);
////	hipFree(d_C);
////
////	// Destroy the handle
////	hipblasDestroy(handle);
////
////	// Free CPU memory
////	free(h_A);
////	free(h_B);
////	free(h_C);
////
////	return 0;
////}
//
//
//#include <stdio.h>
//#include <stdlib.h>
//#include <cstdlib>
//#include <hiprand.h>
//#include <assert.h>
//#include <unistd.h>
//#include <hipblas.h>
//#include <iostream>
//#include <complex.h>
//#include <math.h>
//#include <hip/hip_complex.h>
//#include <hip/hip_runtime.h>
//#include "cublas_beamformer.h"
//
//using namespace std;
//
//// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
//void GPU_fill(hipComplex *A, int nr_rows_A, int nr_cols_A) {
//	hipComplex *G;
//	G = new hipComplex[nr_rows_A*nr_cols_A];
//	for(int i = 0; i < nr_rows_A*nr_cols_A; ++i){
//		G[i].x = (i + 1)%(nr_rows_A*nr_cols_A/(N_BIN));
//		G[i].y = (i + 1)%(nr_rows_A*nr_cols_A/(N_BIN));
//
//	}
//
//	hipMemcpy(A,G,nr_rows_A * nr_cols_A * sizeof(hipComplex),hipMemcpyHostToDevice);
//	delete[] G;
//}
//
//void GPU_fill2(hipComplex *A, int nr_rows_A, int nr_cols_A) {
//	hipComplex *G;
//	G = new hipComplex[nr_rows_A*nr_cols_A];
//	for(int i = 0; i < nr_rows_A*nr_cols_A; ++i){
//		G[i].x = i%(nr_rows_A*nr_cols_A/(N_BIN));
//		G[i].y = i%(nr_rows_A*nr_cols_A/(N_BIN));
//	}
//
//	hipMemcpy(A,G,nr_rows_A * nr_cols_A * sizeof(hipComplex),hipMemcpyHostToDevice);
//	delete[] G;
//}
//
//void print_matrix(const hipComplex *A, int nr_rows_A, int nr_cols_A, int nr_sheets_A) {
//	for(int i = 0; i < nr_rows_A; ++i){
//		for(int j = 0; j < nr_cols_A; ++j){
//			for(int k = 0; k < nr_sheets_A; ++k){
//				//				cout << A[j * nr_rows_A + i].x << "+" << A[j * nr_rows_A + i].y << "i" <<" ";
//				printf("%i,%i,%i: %e + %e i\n",i,j,k,A[k*nr_rows_A*nr_cols_A + j * nr_rows_A + i].x, A[k*nr_rows_A*nr_cols_A + j * nr_rows_A + i].y);
//			}
//		}
//		//			cout << endl;
//	}
//	//		cout << endl;
//	//	for(int i = 0; i < nr_rows_A*nr_cols_A; ++i){
//	//		printf("%i,: %e + %e i\n",i,A[i].x, A[i].y);
//	//	}
//}
//
//
//void print_matrix2(const float *A, int nr_rows_A, int nr_cols_A) {
//	//	for(int j = 0; j < nr_cols_A; ++j){
//	//		for(int i = 0; i < nr_rows_A; ++i){
//	//			//cout << A[j * nr_rows_A + i].x << "+" << A[j * nr_rows_A + i].y << "i" <<" ";
//	//			printf("%i,%i: %e\n",i,j,A[j * nr_rows_A + i]);
//	//		}
//	//		cout << endl;
//	//	}
//	//	cout << endl;
//
//	for(int i = 0; i < nr_rows_A*nr_cols_A; ++i){
//		printf("%i,: %e\n",i,A[i]);
//	}
//}
//
//__global__
//void data_restructure(hipComplex * data, hipComplex * data_restruc){
//
//	int e = threadIdx.x;
//	int t = blockIdx.x;
//	int f = blockIdx.y;
//
//	data_restruc[f*N_TIME*N_ELE + t*N_ELE + e] = data[t*N_BIN*N_ELE + f*N_ELE + e];
//}
//
//
//void beamform(const hipComplex * d_A,
//		const hipComplex * d_B,hipblasHandle_t handle,
//		hipComplex * d_C) {
//
//	// Allocate 3 arrays on CPU
//	hipError_t cudaStat;
//
//	int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;
//
//	nr_rows_A = N_BEAM;
//	nr_cols_A = N_ELE;
//	nr_rows_B = N_ELE;
//	nr_cols_B = N_TIME;
//	nr_rows_C = N_BEAM;
//	nr_cols_C = N_TIME;
//
//	const hipComplex **h_arr_A = 0; const hipComplex **h_arr_B = 0; hipComplex **h_arr_C = 0;
//	//New code ////////
//	h_arr_A = (const hipComplex **)malloc(nr_rows_A * nr_cols_A *N_BIN*sizeof(const hipComplex*)); //N_TIME instead of N_BIN
//	h_arr_B = (const hipComplex **)malloc(nr_rows_B * nr_cols_B *N_BIN*sizeof(const hipComplex*)); //N_TIME instead of N_BIN
//	h_arr_C = (hipComplex **)malloc(nr_rows_C * nr_cols_C *N_BIN*sizeof(hipComplex*)); //N_TIME instead of N_BIN
//
//	for(int i = 0; i < N_BIN; i++){ //N_TIME instead of N_BIN
//		h_arr_A[i] = d_A + i*nr_rows_A*nr_cols_A;
//		h_arr_B[i] = d_B + i*nr_rows_B*nr_cols_B;
//		h_arr_C[i] = d_C + i*nr_rows_C*nr_cols_C;
//	}
//
//	//	delete[] d_A;
//	//	delete[] d_B;
//
//	hipComplex **d_arr_A = 0; hipComplex **d_arr_B = 0; hipComplex **d_arr_C = 0;
//	cudaStat = hipMalloc((void **)&d_arr_A,nr_rows_A * nr_cols_A * N_BIN * sizeof(hipComplex*)); //N_TIME instead of N_BIN
//	assert(!cudaStat);
//	cudaStat = hipMalloc((void **)&d_arr_B,nr_rows_B * nr_cols_B * N_BIN * sizeof(hipComplex*)); //N_TIME instead of N_BIN
//	assert(!cudaStat);
//	cudaStat = hipMalloc((void **)&d_arr_C,nr_rows_C * nr_cols_C * N_BIN * sizeof(hipComplex*)); //N_TIME instead of N_BIN
//	assert(!cudaStat);
//
//	cudaStat = hipMemcpy(d_arr_A,h_arr_A,nr_rows_A * nr_cols_A * N_BIN * sizeof(hipComplex*),hipMemcpyHostToDevice); //N_TIME instead of N_BIN
//	assert(!cudaStat);
//	cudaStat = hipMemcpy(d_arr_B,h_arr_B,nr_rows_B * nr_cols_B * N_BIN * sizeof(hipComplex*),hipMemcpyHostToDevice); //N_TIME instead of N_BIN
//	assert(!cudaStat);
//	cudaStat = hipMemcpy(d_arr_C,h_arr_C,nr_rows_C * nr_cols_C * N_BIN * sizeof(hipComplex*),hipMemcpyHostToDevice); //N_TIME instead of N_BIN
//	assert(!cudaStat);
//
//	int lda=nr_rows_A,ldb=nr_rows_B,ldc=nr_rows_C;
//	hipComplex alf;
//	hipComplex bet;
//
//	alf.x = 1;
//	alf.y = 0;
//	bet.x = 0;
//	bet.y = 0;
//	//New variables
//	int batchCount = N_BIN;
//
//	// Create a handle for CUBLAS
//	hipblasCreate(&handle);
//
//	hipblasStatus_t stat;
//
//	stat = hipblasCgemmBatched(
//			handle,
//			HIPBLAS_OP_N, // HIPBLAS_OP_N,
//			HIPBLAS_OP_N,
//			nr_rows_A,
//			nr_cols_B,
//			nr_cols_A,
//			&alf,
//			(const hipComplex **)d_arr_A,
//			lda,
//			(const hipComplex **)d_arr_B,
//			ldb,
//			&bet,
//			(hipComplex **)d_arr_C,
//			ldc,
//			batchCount);
//
//
//	if(stat != HIPBLAS_STATUS_SUCCESS){
//		cerr << "hipblasCgemmBatched failed" << endl;
//		exit(1);
//	}
//	assert(!hipGetLastError());
//
//	//////////////////////
//
//	//Free GPU memory
//	//	hipFree(d_A);
//	//	hipFree(d_B);
//	//	hipFree(d_C);
//
//	// Destroy the handle
//	//hipblasDestroy(handle);
//
//}
//
//__global__
//void sti_reduction(const hipComplex * beamformed,
//		float * data_out) {
//
//	int f = blockIdx.x;
//	int b = blockIdx.y;
//	int t = threadIdx.x;
//	int s = blockIdx.z;
//
//	int h = sample_idx(s*N_TIME_STI + t,b,f);
//	int h1 = sample_idx(s*N_TIME_STI + t,b+N_BEAM1,f);
//
//	float beam_power1;
//	float beam_power2;
//	float cross_power1;
//	float cross_power2;
//
//	hipFloatComplex samp1;
//	hipFloatComplex samp2;
//	float scale = 1.0/N_TIME_STI;
//
//	//New variables//////
//	__shared__ hipFloatComplex reduced_array1[N_STI_BLOC];
//	__shared__ hipFloatComplex reduced_array[N_STI_BLOC];
//	/////////////////////
//
//	if (t < N_TIME_STI) {
//		samp1.x = beamformed[h].x;
//		samp1.y = beamformed[h].y;
//		beam_power1 = (samp1.x * samp1.x) + (samp1.y * samp1.y);
//		reduced_array[t].x = beam_power1;
//
//		samp2.x = beamformed[h1].x;
//		samp2.y = beamformed[h1].y;
//		beam_power2 = (samp2.x * samp2.x) + (samp2.y * samp2.y);
//		reduced_array[t].y = beam_power2;
//
//		cross_power1 = (samp1.x * samp2.x) + (samp1.y * samp2.y);
//		cross_power2 = (samp1.y * samp2.x) - (samp1.x * samp2.y);
//		reduced_array1[t].x = cross_power1;
//		reduced_array1[t].y = cross_power2;
//	}
//
//	//New code///////////////////////////////////////////////
//	else{
//		reduced_array[t].x = 0.0;
//		reduced_array[t].y = 0.0;
//		reduced_array1[t].x = 0.0;
//		reduced_array1[t].y = 0.0;
//	}
//	__syncthreads();
//
//	for(int k = blockDim.x/2; k>0; k>>=1){
//		if(t<k){
//			reduced_array[t].x += reduced_array[t+k].x;
//			reduced_array[t].y += reduced_array[t+k].y;
//			reduced_array1[t].x += reduced_array1[t+k].x;
//			reduced_array1[t].y += reduced_array1[t+k].y;
//		}
//		__syncthreads();
//	}
//
//	if(t == 0){
//		//New Code
//		data_out[output_idx(0,b,s,f)] = reduced_array[0].x*scale; //x pol
//		data_out[output_idx(1,b,s,f)] = reduced_array[0].y*scale; //y pol
//		data_out[output_idx(2,b,s,f)] = reduced_array1[0].x*scale; //cross pol (x)
//		data_out[output_idx(3,b,s,f)] = reduced_array1[0].y*scale;//cross pol (y)
//	}
//}





#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include <hiprand.h>
#include <assert.h>
#include <unistd.h>
#include <hipblas.h>
#include <iostream>
#include <complex.h>
#include <math.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include "cublas_beamformer.h"

using namespace std;

// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
void GPU_fill(hipComplex *A, int nr_rows_A, int nr_cols_A) {
	hipComplex *G;
	G = new hipComplex[nr_rows_A*nr_cols_A];
	for(int i = 0; i < nr_rows_A*nr_cols_A; ++i){
		G[i].x = (i + 1)%(nr_rows_A*nr_cols_A/(N_BIN));
		G[i].y = (i + 1)%(nr_rows_A*nr_cols_A/(N_BIN));

	}

	hipMemcpy(A,G,nr_rows_A * nr_cols_A * sizeof(hipComplex),hipMemcpyHostToDevice);
	delete[] G;
}

void GPU_fill2(hipComplex *A, int nr_rows_A, int nr_cols_A) {
	hipComplex *G;
	G = new hipComplex[nr_rows_A*nr_cols_A];
	for(int i = 0; i < nr_rows_A*nr_cols_A; ++i){
		G[i].x = i%(nr_rows_A*nr_cols_A/(N_BIN));
		G[i].y = i%(nr_rows_A*nr_cols_A/(N_BIN));
	}

	hipMemcpy(A,G,nr_rows_A * nr_cols_A * sizeof(hipComplex),hipMemcpyHostToDevice);
	delete[] G;
}

void print_matrix(const hipComplex *A, int nr_rows_A, int nr_cols_A, int nr_sheets_A) {
	for(int i = 0; i < nr_rows_A; ++i){
		for(int j = 0; j < nr_cols_A; ++j){
			for(int k = 0; k < nr_sheets_A; ++k){
				//				cout << A[j * nr_rows_A + i].x << "+" << A[j * nr_rows_A + i].y << "i" <<" ";
				printf("%i,%i,%i: %e + %e i\n",i,j,k,A[k*nr_rows_A*nr_cols_A + j * nr_rows_A + i].x, A[k*nr_rows_A*nr_cols_A + j * nr_rows_A + i].y);
			}
		}
		//			cout << endl;
	}
	//		cout << endl;
	//	for(int i = 0; i < nr_rows_A*nr_cols_A; ++i){
	//		printf("%i,: %e + %e i\n",i,A[i].x, A[i].y);
	//	}
}


void print_matrix2(const float *A, int nr_rows_A, int nr_cols_A) {
	//	for(int j = 0; j < nr_cols_A; ++j){
	//		for(int i = 0; i < nr_rows_A; ++i){
	//			//cout << A[j * nr_rows_A + i].x << "+" << A[j * nr_rows_A + i].y << "i" <<" ";
	//			printf("%i,%i: %e\n",i,j,A[j * nr_rows_A + i]);
	//		}
	//		cout << endl;
	//	}
	//	cout << endl;

	for(int i = 0; i < nr_rows_A*nr_cols_A; ++i){
		printf("%i,: %e\n",i,A[i]);
	}
}

static hipComplex * d_weights = NULL;
void update_weights(char * filename){
	char weight_filename[128];
	strcpy(weight_filename, filename);
	FILE * weights;
	float * bf_weights;
	float complex * weights_dc;
	float complex * weights_dc_n;

	// Allocate heap memory for file data
	bf_weights = (float *)malloc(2*N_WEIGHTS*sizeof(float));
	weights_dc = (float complex *)malloc(N_WEIGHTS*sizeof(float complex *));
	weights_dc_n = (float complex *)malloc(N_WEIGHTS*sizeof(float complex *));
	weights = fopen(weight_filename, "r");

	int j;
	if (weights != NULL) {
		fread(bf_weights, sizeof(float), 2*N_WEIGHTS, weights);

		// Convert to complex numbers (do a conjugate at the same time)
		for(j = 0; j < N_WEIGHTS; j++){
			weights_dc_n[j] = bf_weights[2*j] - bf_weights[(2*j)+1]*I;
		}

		// Transpose the weights
		int m,n;
		float complex transpose[N_BEAM][N_ELE*N_BIN];
		for(m=0;m<N_BEAM;m++){
			for(n=0;n<N_ELE*N_BIN;n++){
				transpose[m][n] = weights_dc_n[m*N_ELE*N_BIN + n];
			}
		}
		for(n=0;n<N_ELE*N_BIN;n++){
			for(m=0;m<N_BEAM;m++){
				weights_dc[n*N_BEAM+ m] = transpose[m][n];
			}
		}
		fclose(weights);
	}
	free(bf_weights);

	// Copy weights to device
	hipMemcpy(d_weights, weights_dc, N_WEIGHTS*sizeof(hipComplex), hipMemcpyHostToDevice); //r_weights instead of weights_dc //*N_TIME

	free(weights_dc);
}

static hipComplex **d_arr_A = NULL; static hipComplex **d_arr_B = NULL; static hipComplex **d_arr_C = NULL;
static hipComplex * d_beamformed = NULL;
static hipComplex * d_data = NULL;
static hipComplex * d_data1 = NULL;
static float * d_outputs;

void init_beamformer(){
	// Allocate memory for the weights, data, beamformer output, and sti output.

	hipMalloc((void **)&d_weights, N_WEIGHTS*sizeof(hipComplex)); //*N_TIME

	hipMalloc((void **)&d_data1, N_SAMP*sizeof(hipComplex));

	hipMalloc((void **)&d_data, N_SAMP*sizeof(hipComplex));

	hipError_t err_malloc = hipMalloc((void **)&d_beamformed, N_TBF*sizeof(hipComplex));
	if (err_malloc != hipSuccess) {
		printf("CUDA Error (cudaMalloc2): %s\n", hipGetErrorString(err_malloc));
	}

	hipMalloc((void **)&d_outputs, N_POL*(N_OUTPUTS*sizeof(float)/2));

	// This is all memory allocated to arrays that are used by gemmBatched.
	// Allocate 3 arrays on CPU
	hipError_t cudaStat;

	int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

	nr_rows_A = N_BEAM;
	nr_cols_A = N_ELE;
	nr_rows_B = N_ELE;
	nr_cols_B = N_TIME;
	nr_rows_C = N_BEAM;
	nr_cols_C = N_TIME;

	// Allocate memory to host arrays.
	const hipComplex **h_arr_A = 0; const hipComplex **h_arr_B = 0; hipComplex **h_arr_C = 0;
	h_arr_A = (const hipComplex **)malloc(nr_rows_A * nr_cols_A *N_BIN*sizeof(const hipComplex*));
	h_arr_B = (const hipComplex **)malloc(nr_rows_B * nr_cols_B *N_BIN*sizeof(const hipComplex*));
	h_arr_C = (hipComplex **)malloc(nr_rows_C * nr_cols_C *N_BIN*sizeof(hipComplex*));

	// Allocate memory for each batch in an array.
	for(int i = 0; i < N_BIN; i++){
		h_arr_A[i] = d_weights + i*nr_rows_A*nr_cols_A;
		h_arr_B[i] = d_data + i*nr_rows_B*nr_cols_B;
		h_arr_C[i] = d_beamformed + i*nr_rows_C*nr_cols_C;
	}

	//	delete[] d_A;
	//	delete[] d_B;

	// Allocate memory to arrays on device.
	cudaStat = hipMalloc((void **)&d_arr_A,nr_rows_A * nr_cols_A * N_BIN * sizeof(hipComplex*));
	assert(!cudaStat);
	cudaStat = hipMalloc((void **)&d_arr_B,nr_rows_B * nr_cols_B * N_BIN * sizeof(hipComplex*));
	assert(!cudaStat);
	cudaStat = hipMalloc((void **)&d_arr_C,nr_rows_C * nr_cols_C * N_BIN * sizeof(hipComplex*));
	assert(!cudaStat);

	// Copy memory from host to device.
	cudaStat = hipMemcpy(d_arr_A,h_arr_A,nr_rows_A * nr_cols_A * N_BIN * sizeof(hipComplex*),hipMemcpyHostToDevice);
	assert(!cudaStat);
	cudaStat = hipMemcpy(d_arr_B,h_arr_B,nr_rows_B * nr_cols_B * N_BIN * sizeof(hipComplex*),hipMemcpyHostToDevice);
	assert(!cudaStat);
	cudaStat = hipMemcpy(d_arr_C,h_arr_C,nr_rows_C * nr_cols_C * N_BIN * sizeof(hipComplex*),hipMemcpyHostToDevice);
	assert(!cudaStat);

}

__global__
void data_restructure(hipComplex * data, hipComplex * data_restruc){

	int e = threadIdx.x;
	int t = blockIdx.x;
	int f = blockIdx.y;

	//Restructure data so that the frequency bin is the slowest moving index
	data_restruc[f*N_TIME*N_ELE + t*N_ELE + e] = data[t*N_BIN*N_ELE + f*N_ELE + e];
}

void data_in(char * input_filename){
	FILE * data;

	// File data pointers
	float * bf_data;

	// Complex data pointers
	float complex * data_dc;

	// Allocate heap memory for file data
	bf_data = (float *)malloc(2*N_SAMP*sizeof(float));
	data_dc = (float complex *)malloc(N_SAMP*sizeof(float complex *));

	// Open files
	data = fopen(input_filename, "r");

	/*********************************************************
	 * Read in Data
	 *********************************************************/
	if (data != NULL) {
		fread(bf_data, sizeof(float), 2*N_SAMP, data);
		int j;
		// Make 'em complex!
		for (j = 0; j < N_SAMP; j++) {
			data_dc[j] = bf_data[2*j] + bf_data[(2*j)+1]*I;
		}

		// Specify grid and block dimensions
		dim3 dimBlock_d(N_ELE, 1, 1);
		dim3 dimGrid_d(N_TIME, N_BIN, 1);

		hipComplex * d_data_in = d_data1;
		hipComplex * d_data_out = d_data;

		hipMemcpy(d_data_in,    data_dc,   N_SAMP*sizeof(hipComplex), hipMemcpyHostToDevice);

		// Restructure data for hipblasCgemmBatched function.
		data_restructure<<<dimGrid_d, dimBlock_d>>>(d_data_in, d_data_out);

		fclose(data);
	}
	free(bf_data);
	free(data_dc);
}

void beamform(hipblasHandle_t handle) {
	int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C;
	nr_rows_A = N_BEAM;
	nr_cols_A = N_ELE;
	nr_rows_B = N_ELE;
	nr_cols_B = N_TIME;
	nr_rows_C = N_BEAM;

	// Leading dimensions are always the rows of each matrix since the data is stored in a column-wise order.
	int lda=nr_rows_A,ldb=nr_rows_B,ldc=nr_rows_C;
	hipComplex alf;
	hipComplex bet;

	alf.x = 1;
	alf.y = 0;
	bet.x = 0;
	bet.y = 0;

	int batchCount = N_BIN; 				// There must be the same number of batches in each array.

	hipblasStatus_t stat;
	/*
		This function performs a matrix multiplication of the data and the weights.
		Weights - d_arr_A, Data - d_arr_B, and the output - d_arr_C.
	*/
	stat = hipblasCgemmBatched(
			handle,							// handle to the cuBLAS library context.
			HIPBLAS_OP_N,					// Operation on matrices within array A.
			HIPBLAS_OP_N,					// Operation on matrices within array B.
			nr_rows_A,						// Number of rows in matrix A and C.
			nr_cols_B,						// Number of columns in matrix B and C.
			nr_cols_A,						// Number of columns and rows in matrix A and B respectively.
			&alf,							// Scalar used for multiplication.
			(const hipComplex **)d_arr_A,	// Weight array of pointers.
			lda,							// Leading dimension of each batch or matrix in array A.
			(const hipComplex **)d_arr_B,	// Data array of pointers.
			ldb,							// Leading dimension of each batch or matrix in array B.
			&bet,							// Scalar used for multiplication.
			(hipComplex **)d_arr_C,			// Output array of pointers.
			ldc,							// Leading dimension of each batch or matrix in array C.
			batchCount);					// Number of batches in each array.


	if(stat != HIPBLAS_STATUS_SUCCESS){
		cerr << "hipblasCgemmBatched failed" << endl;
		exit(1);
	}
	assert(!hipGetLastError());

	//Free GPU memory
	//	hipFree(d_A);
	//	hipFree(d_B);
	//	hipFree(d_C);

	// Destroy the handle
	//hipblasDestroy(handle);

}

__global__
void sti_reduction(hipComplex * data_in, float * data_out) {

	int f = blockIdx.x;
	int b = blockIdx.y;
	int t = threadIdx.x;
	int s = blockIdx.z;

	int h = sample_idx(s*N_TIME_STI + t,b,f);						// Preprocessor macro used for the output of the beamformer. More detail can be seen in the header file. (First set of beams)
	int h1 = sample_idx(s*N_TIME_STI + t,b+N_BEAM1,f);				// Preprocessor macro used for the output of the beamformer. More detail can be seen in the header file. (Last set of beams)

	// Temporary variables used for updating.
	float beam_power1;
	float beam_power2;
	float cross_power1;
	float cross_power2;

	hipFloatComplex samp1;
	hipFloatComplex samp2;
	float scale = 1.0/N_TIME_STI; 									// Scale power by number of samples per STI window.

	__shared__ hipFloatComplex reduced_array1[N_STI_BLOC];
	__shared__ hipFloatComplex reduced_array[N_STI_BLOC];

	if (t < N_TIME_STI) {
		// X polarization (XX*).
		samp1.x = data_in[h].x;
		samp1.y = data_in[h].y;
		beam_power1 = (samp1.x * samp1.x) + (samp1.y * samp1.y);	// Beamformer output multiplied by its conjugate (absolute value squared).
		reduced_array[t].x = beam_power1;

		// Y polarization (YY*).
		samp2.x = data_in[h1].x;
		samp2.y = data_in[h1].y;
		beam_power2 = (samp2.x * samp2.x) + (samp2.y * samp2.y);	// Beamformer output multiplied by its conjugate (absolute value squared).
		reduced_array[t].y = beam_power2;

		// Cross polarization (XY*).
		cross_power1 = (samp1.x * samp2.x) + (samp1.y * samp2.y);	// Real part of cross polarization.
		cross_power2 = (samp1.y * samp2.x) - (samp1.x * samp2.y);	// Imaginary part of cross polarization.
		reduced_array1[t].x = cross_power1;
		reduced_array1[t].y = cross_power2;
	}
	else{
		reduced_array[t].x = 0.0;
		reduced_array[t].y = 0.0;
		reduced_array1[t].x = 0.0;
		reduced_array1[t].y = 0.0;
	}
	__syncthreads();

	// Reduction is performed by splitting up the threads in each block and summing them all up.
	// The number of threads in each block needs to be a power of two in order for the reduction to work. (No left over threads).
	for(int k = blockDim.x/2; k>0; k>>=1){
		if(t<k){
			reduced_array[t].x += reduced_array[t+k].x;
			reduced_array[t].y += reduced_array[t+k].y;
			reduced_array1[t].x += reduced_array1[t+k].x;
			reduced_array1[t].y += reduced_array1[t+k].y;
		}
		__syncthreads();
	}

	// After reduction is complete, assign each reduced to value to appropriate position in output array.
	if(t == 0){
		data_out[output_idx(0,b,s,f)] = reduced_array[0].x*scale; 	// XX*.
		data_out[output_idx(1,b,s,f)] = reduced_array[0].y*scale; 	// YY*.
		data_out[output_idx(2,b,s,f)] = reduced_array1[0].x*scale; 	// XY* real.
		data_out[output_idx(3,b,s,f)] = reduced_array1[0].y*scale;	// XY* imaginary.
	}
}

void run_beamformer(hipblasHandle_t handle, float * data_out){
	// Specify grid and block dimensions
	dim3 dimBlock(N_STI_BLOC, 1, 1);
	dim3 dimGrid(N_BIN, N_BEAM1, N_STI);

	printf("Starting beamformer\n");

	// Call beamformer function containing hipblasCgemmBatched()
	beamform(handle);
	hipError_t err_code = hipGetLastError();
	if (err_code != hipSuccess) {
		printf("CUDA Error (beamform): %s\n", hipGetErrorString(err_code));
	}

	hipComplex * d_data_in = d_beamformed;
	float * d_data_out = d_outputs;

	printf("Starting sti_reduction\n");

	// Call STI reduction kernel.
	sti_reduction<<<dimGrid, dimBlock>>>(d_data_in, d_data_out);

	printf("Finishing sti_reduction\n");

	err_code = hipGetLastError();
	if (err_code != hipSuccess) {
		printf("CUDA Error (sti_reduction): %s\n", hipGetErrorString(err_code));
	}

	// Copy output data from device to host.
	hipMemcpy(data_out, d_data_out, N_POL*(N_OUTPUTS*sizeof(float)/2),hipMemcpyDeviceToHost);

	hipFree(d_data);
	hipFree(d_outputs);
}
